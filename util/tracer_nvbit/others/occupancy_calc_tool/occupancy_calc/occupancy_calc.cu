/* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Author: Oreste Villa, ovilla@nvidia.com - 2018 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* provide some __device__ functions */
#include "utils/utils.h"

/* kernel id counter, maintained in system memory */
uint32_t kernel_id = 0;

/* total instruction counter, maintained in system memory, incremented by
 * "counter" every time a kernel completes  */
uint64_t tot_app_instrs = 0;

/* kernel instruction counter, updated by the GPU */
__managed__ uint64_t counter = 0;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
uint32_t ker_begin_interval = 0;
uint32_t ker_end_interval = UINT32_MAX;
int verbose = 0;
int count_warp_level = 1;
int exclude_pred_off = 0;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

bool skip_flag = false;
bool alternate = false;

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
void nvbit_at_init() {
  /* just make sure all managed variables are allocated on GPU */
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

  /* we get some environment variables that are going to be use to selectively
   * instrument (within a interval of kernel indexes and instructions). By
   * default we instrument everything. */
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
              "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(ker_begin_interval, "KERNEL_BEGIN", 0,
              "Beginning of the kernel launch interval where to apply "
              "instrumentation");
  GET_VAR_INT(
      ker_end_interval, "KERNEL_END", UINT32_MAX,
      "End of the kernel launch interval where to apply instrumentation");
  GET_VAR_INT(count_warp_level, "COUNT_WARP_LEVEL", 1,
              "Count warp level or thread level instructions");
  GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 0,
              "Exclude predicated off instruction from count");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  std::string pad(100, '-');
  printf("%s\n", pad.c_str());
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
  if (skip_flag)
    return;
  /* Identify all the possible CUDA launch events */
  if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
      cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
      cbid == API_CUDA_cuLaunchKernel) {
    /* cast params to cuLaunch_params since if we are here we know these are
     * the right parameters type */
    cuLaunch_params *p = (cuLaunch_params *)params;
    cuLaunchKernel_params_st *p_2 = (cuLaunchKernel_params_st *)params;

    // Print out the occupancy for each kernel
    if (alternate) {
      alternate = false;
      int blocks;
      int threads = p_2->blockDimX * p_2->blockDimY * p_2->blockDimZ;
      CUDA_SAFECALL(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(
          &blocks, p->f, threads, p_2->sharedMemBytes));
      printf("Kernel = %s\n", nvbit_get_func_name(ctx, p->f));
      // Warning! Only for volta with 80 SMs
      printf("Max Blocks / GPU = %d\n", blocks * 80);
      skip_flag = false;
    } else {
      alternate = true;
    }
    if (!is_exit) {
      /* if we are entering in a kernel launch:
       * 1. Lock the mutex to prevent multiple kernels to run concurrently
       * (overriding the counter) in case the user application does that
       * 2. Select if we want to run the instrumented or original
       * version of the kernel
       * 3. Reset the kernel instruction counter */

      pthread_mutex_lock(&mutex);
    } else {
      /* if we are exiting a kernel launch:
       * 1. Wait until the kernel is completed using
       * hipDeviceSynchronize()
       * 2. Get number of thread blocks in the kernel
       * 3. Print the thread instruction counters
       * 4. Release the lock*/
      CUDA_SAFECALL(hipDeviceSynchronize());
      pthread_mutex_unlock(&mutex);
    }
  }
}
